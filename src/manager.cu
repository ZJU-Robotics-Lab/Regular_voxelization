/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <chrono>
#include <thread>
using namespace std;

GPUTransformer::GPUTransformer (float* point_host_, int size_, int* x_, int* y_, int* height_, int max_length_, int num_x_, int num_y_, int num_height_, int outsize_) {
  point_host = point_host_;
  h_max_length = max_length_;
  h_num_x = num_x_;
  h_num_height = num_height_;
  h_num_y = num_y_;
  outsize = outsize_;

  size = size_* 3 * sizeof(float);
  d_size = size_;
  int grid_size = num_x_ * num_y_ * num_height_ * outsize * sizeof(int);
  d_grid_size = num_x_ * num_y_ * num_height_ ;

  hipMalloc((void**) &x, d_size * sizeof(int));
  hipMalloc((void**) &y, d_size * sizeof(int));
  hipMalloc((void**) &height, d_size * sizeof(int));

  // auto t1 = std::chrono::high_resolution_clock::now();
  hipMalloc((void**) &point_device, size);

  // printf("err0 %s\n",cudaGetErrorStx(err));

  // auto t2 = std::chrono::high_resolution_clock::now();
  // std::cout << "memcpy took "
  //           << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count()
  //           << " milliseconds\n";

  hipMemcpy(point_device, point_host, size, hipMemcpyHostToDevice);
  hipMemcpy(y, y_, d_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(height, height_, d_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(x, x_, d_size * sizeof(int), hipMemcpyHostToDevice);
}

void GPUTransformer::transform() {
  dim3 blockSize(256);
  dim3 gridSize((d_size + blockSize.x - 1) / blockSize.x);
  point2gridmap<<<gridSize, blockSize>>>(point_device, x, y, height, d_size, h_max_length, h_num_x, h_num_y, h_num_height);
  hipDeviceSynchronize();
}

void GPUTransformer::retreive(float* point_transformed) {

  int pt_count = 0;
  int index = 0;
  int x_h[d_size] = {0};
  int y_h[d_size] = {0};
  int height_h[d_size] = {0};
  int counter[d_grid_size] = {0};
  int total = 0;
  int tmp[d_grid_size*10] = {0};

  hipMemcpy(x_h, x, d_size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(y_h, y, d_size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(height_h, height, d_size * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < d_size; i++)
  {
    if(counter[y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x] < 10 && total < outsize)
    {
      total ++;
      pt_count = counter[y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x];
      tmp[y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x + pt_count * h_num_height * h_num_x * h_num_y] = i;
      counter[y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x] ++;
    }
  }

  int k = 0;
  for (int i = 0; i < (d_grid_size*10); i++)
  { 
    if(tmp[i] != 0)
    {
      point_transformed[3*(k) + 0] = point_host[tmp[i]];
      point_transformed[3*(k) + 1] = point_host[tmp[i]+d_size];
      point_transformed[3*(k) + 2] = point_host[tmp[i]+2*d_size];
      k ++;
    }
  }

  hipFree(point_device);
  hipFree(height);
  hipFree(y);
  hipFree(x);
}

GPUTransformer::~GPUTransformer() {
  hipFree(point_device);
  hipFree(height);
  hipFree(y);
  hipFree(x);
}
